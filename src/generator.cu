#include "hip/hip_runtime.h"
#include "./chunk_dOOm_gen.hpp"
#include "./cuda_noise.cuh"
#include "./generator.hpp"
#include <algorithm>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

// Device function to generate a single ChunkSmol
__device__ void generateSmolChunk(ChunkSmol *chunk_smol, int32_t i_ch,
                                  const float *heights) {
    for (uint32_t i_y = 0; i_y < 16; i_y++) {
        // Process all 4096 blocks in this subchunk
        for (uint32_t i_z = 0; i_z < 16; i_z++) {
            for (uint32_t i_x = 0; i_x < 16; i_x++) {
                // Get height value from device vector
                float height = heights[i_z * 16 + i_x];

                // Calculate absolute Y coordinate in the world
                int32_t absolute_y = i_y + i_ch * 16;

                // Set the appropriate block based on height
                if (height > absolute_y) {
                    chunk_smol->setBlock(i_y, i_z, 15 - i_x,
                                         make_block("minecraft:stone"));
                } else {
                    chunk_smol->setBlock(i_y, i_z, 15 - i_x,
                                         make_block("minecraft:air"));
                }
            }
        }
    }
}

std::vector<Chunk> ChunkGenerator::generate_all(int32_t region_x,
                                                int32_t region_z) {
    std::vector<Chunk> chunks;
    chunks.reserve(32 * 32);

    thrust::device_vector<float> heights(16 * 16 * 32 * 32);

    // Use a lambda to map from index to (x, z) and call
    // getBaseTerrainHeight
    thrust::transform(
        thrust::counting_iterator<uint32_t>(0),
        thrust::counting_iterator<uint32_t>(16 * 16 * 32 * 32), heights.begin(),
        [region_x, region_z] __device__(uint32_t idx) {
            // Array of [[chunk]] where chunk is locally indexed
            int32_t cell_id = idx / (16 * 16);

            int32_t cell_id_x = cell_id / 32;
            int32_t cell_id_z = cell_id % 32;

            int32_t local_x = idx % 16;
            int32_t local_z = (idx / 16) % 16;

            int32_t chunk_x = (region_x * 32 + cell_id_x) * 16;
            int32_t chunk_z = (region_z * 32 + cell_id_z) * 16;

            return getBaseTerrainHeight(chunk_x + local_x, chunk_z + local_z);
        });

    thrust::device_vector<ChunkSmol *> chunk_smols(32 * 32);

    for (auto cell_id = 0; cell_id < 32 * 32; cell_id++) {
        int32_t cell_id_x = cell_id / 32;
        int32_t cell_id_z = cell_id % 32;
        float *cell_heights = thrust::raw_pointer_cast(heights.data()) +
                              (cell_id_x * 16 * 16 * 32 + cell_id_z * 16 * 16);

        int32_t x = (region_x * 32 + cell_id_x) * 16;
        int32_t z = (region_z * 32 + cell_id_z) * 16;

        auto chunk = &chunks.emplace_back(x, z);
        chunk_smols[cell_id] = chunk->chunk_smols.get();
    }

    ChunkSmol **all_chunks = thrust::raw_pointer_cast(chunk_smols.data());
    float *all_heights = thrust::raw_pointer_cast(heights.data());

    thrust::for_each(
        thrust::counting_iterator<uint32_t>(0),
        thrust::counting_iterator<uint32_t>(24 * 32 * 32),
        [all_chunks, all_heights] __device__(const uint32_t &smol_idx) {
            const uint32_t i_ch = smol_idx % 24;
            const uint32_t cell_id = smol_idx / 24;

            ChunkSmol *raw_chunks = all_chunks[cell_id];

            // Placement new to construct the ChunkSmol object
            new (raw_chunks + i_ch) ChunkSmol();

            // Generate this smol chunk
            generateSmolChunk(raw_chunks + i_ch, i_ch,
                              all_heights + cell_id * 16 * 16);
        });

    return chunks;
}

__device__ float ChunkGenerator::getBaseTerrainHeight(float x, float z) {
    auto noise = cudaNoise::repeaterSimplex(
        make_float3(x * 0.01f, 0.f, z * 0.01f), 1.0f, 0, 3, 2.0f, 0.5f);

    // Simple height calculation: noise range -1 to 1, convert to 32-160
    // range
    auto height = (noise + 1) * 64 + 32; // 32 - 160

    return std::clamp<double>(height, 0.0, 320.0);
}
