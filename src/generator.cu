#include "hip/hip_runtime.h"
#include "./chunk_dOOm_gen.hpp"
#include "./cuda_noise.cuh"
#include "./generator.hpp"
#include "block_template.hpp"
#include <algorithm>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

__device__ float noise(int32_t seed, int32_t x, int32_t z, float frequency,
                       int32_t octaves) {
    return cudaNoise::repeaterSimplex(make_float3(x, 0.f, z), frequency, seed,
                                      octaves, 2.0f, 0.5f);
}

__device__ float noise(int32_t seed, int32_t x, int32_t y, int32_t z,
                       float frequency, int32_t octaves) {
    return cudaNoise::repeaterSimplex(make_float3(x, y, z), frequency, seed,
                                      octaves, 2.0f, 0.5f);
}

__device__ bool cave_noise(int32_t seed, int32_t x, int32_t y, int32_t z) {
    float noodle1 = noise(seed, x, y, z, 0.01f, 2);
    float noodle2 = noise(seed + 1, x, y, z, 0.02f, 2);

    float cavern = noise(seed + 2, x, y, z, 0.01, 3);

    float noodle1_probability =
        std::max(1.f - std::abs(noodle1 - 0.2f) * 10.0f, 0.f);
    float noodle2_probability =
        std::max(1.f - std::abs(noodle2 - 0.2f) * 10.0f, 0.f);

    float cavern_probability = std::max(0.f, cavern - 0.6f);

    return (noodle1_probability * noodle2_probability + cavern_probability) >
           0.1f;
}

// All of the 2d noises should be computed here
__device__ FlatInfo ChunkGenerator::get_flat_info(int32_t seed, int32_t x,
                                                  int32_t z) {
    auto continentalness =
        std::clamp<float>((noise(seed, x, z, 0.005f, 2) + 1) * 0.5f, 0, 1);

    auto shatter =
        std::clamp<float>(noise(seed + 1, x, z, 0.005f, 2) * 0.5f + 0.5f, 0.f,
                          1.f) *
        continentalness * std::clamp<float>(continentalness * 2 - 0.5, 0, 1);
    auto heightNoise = (noise(seed, x, z, 0.01f, 3) + 2.f + shatter) / 4.f;
    auto height = heightNoise * continentalness * 64.f + 48.f;

    auto temperature = std::clamp<float>(
        noise(seed + 2, x, z, 0.005, 4) * 0.5f + 0.5f, 0.f, 1.f);
    auto vegetation = std::clamp<float>(noise(seed + 3, x, z, 0.25f, 1) * 0.5f +
                                            0.5f + shatter * 0.1f,
                                        0.f, 1.f);

    return FlatInfo{height, shatter, temperature, vegetation, continentalness};
}

// And here we use them
__device__ void
ChunkGenerator::generateSmolChunk(ChunkSmol *chunk_smol, int32_t seed,
                                  int32_t chunk_x, int32_t chunk_y,
                                  int32_t chunk_z, const FlatInfo *flat_info,
                                  const VolumetricInfo *volumetric_info) {
    for (uint32_t local_y = 0; local_y < 16; local_y++) {
        // Process all 4096 blocks in this subchunk
        for (uint32_t local_z = 0; local_z < 16; local_z++) {
            for (uint32_t local_x = 0; local_x < 16; local_x++) {
                int32_t absolute_y = local_y + chunk_y - 64;
                FlatInfo flat = flat_info[local_z * 16 + local_x];
                VolumetricInfo volumetric =
                    volumetric_info[local_y * 16 * 16 + local_z * 16 + local_x];

                auto threshold = std::clamp<float>(absolute_y - flat.height,
                                                   -flat.shatter * 10.f,
                                                   flat.shatter * 10.f) *
                                 0.1f / flat.shatter;

                if (volumetric.density > threshold && !volumetric.cave) {
                    chunk_smol->setBlock(local_y, local_z, 15 - local_x,
                                         make_block("minecraft:stone"));
                } else {
                    chunk_smol->setBlock(local_y, local_z, 15 - local_x,
                                         make_block("minecraft:air"));
                }
            }
        }
    }
}

__device__ void ChunkGenerator::replaceSurface(ChunkWrapper &chunk,
                                               int32_t seed) {
    int32_t surface_heights[16][16];

    for (int32_t local_z = 0; local_z < 16; local_z++) {
        for (int32_t local_x = 0; local_x < 16; local_x++) {
            FlatInfo info = chunk.get_flat_info(local_x, local_z);

            float starting_height = std::max<float>(info.height + 15 + 64,
                                                    64 + 64); // because local

            bool cold = info.temperature < 0.4f;
            bool hit_surface = false;
            surface_heights[local_x][local_z] = 0;

            for (int32_t local_y = starting_height; local_y > 32; local_y--) {
                int32_t absolute_y = local_y - 64;

                if (!surface_heights[local_x][local_z] && absolute_y < 65 &&
                    info.continentalness < 0.75) {
                    if (chunk.isSameBlock(local_y, local_z, 15 - local_x,
                                          make_block("minecraft:air"))) {
                        chunk.setBlock(local_y, local_z, 15 - local_x,
                                       make_block("minecraft:water"));
                    }
                }

                bool air_above =
                    chunk.isSameBlock(local_y + 1, local_z, 15 - local_x,
                                      make_block("minecraft:air"));
                bool water_above =
                    chunk.isSameBlock(local_y + 1, local_z, 15 - local_x,
                                      make_block("minecraft:water"));

                if (cold && air_above &&
                    info.vegetation > (info.temperature - 0.1f) &&
                    chunk.isSameBlock(local_y, local_z, 15 - local_x,
                                      make_block("minecraft:water"))) {
                    chunk.setBlock(local_y, local_z, 15 - local_x,
                                   make_block("minecraft:ice"));
                    continue;
                }

                if (chunk.isSameBlock(local_y, local_z, 15 - local_x,
                                      make_block("minecraft:stone"))) {
                    if (!surface_heights[local_x][local_z]) {
                        surface_heights[local_x][local_z] = local_y;
                    }

                    if (air_above && cold) {
                        chunk.setBlock(local_y + 1, local_z, 15 - local_x,
                                       make_block("minecraft:snow"));
                    }

                    if (info.continentalness < 0.5 || water_above) {
                        if (absolute_y < info.height - 5) {
                            break;
                        }

                        chunk.setBlock(local_y, local_z, 15 - local_x,
                                       make_block("minecraft:sand"));

                        if (info.continentalness < 0.4 &&
                            info.temperature > 0.5f && water_above &&
                            info.vegetation > 0.6 &&
                            surface_heights[local_x][local_z] == local_y) {
                            //  Should grow kelp
                            float max_height =
                                std::min<float>((64.f - absolute_y) / 20.f, 1) *
                                15.f;
                            float height_coef = std::max<float>(
                                (info.vegetation - 0.6) * 10.f, 1);
                            float height = height_coef * max_height;
                            int32_t kelp_top = height + local_y;

                            for (int32_t kelp_y = local_y + 1;
                                 kelp_y < kelp_top; kelp_y++) {
                                if (!chunk.isSameBlock(
                                        kelp_y, local_z, 15 - local_x,
                                        make_block("minecraft:water")))
                                    break;

                                if (kelp_y == kelp_top - 1) {
                                    chunk.setBlock(
                                        kelp_y, local_z, 15 - local_x,
                                        make_block("minecraft:kelp"));
                                } else {
                                    chunk.setBlock(
                                        kelp_y, local_z, 15 - local_x,
                                        make_block("minecraft:kelp_plant"));
                                }
                            }
                        }

                    } else if (air_above) {
                        if (cold) {
                            chunk.setBlock(local_y, local_z, 15 - local_x,
                                           make_block<MAKE_KV("snowy", "true")>(
                                               "minecraft:grass_block"));
                            continue;
                        }

                        chunk.setBlock(local_y, local_z, 15 - local_x,
                                       make_block("minecraft:grass_block"));

                        if (info.vegetation > 0.65) {
                            chunk.setBlock(local_y + 1, local_z, 15 - local_x,
                                           make_block("minecraft:tall_grass"));
                            chunk.setBlock(local_y + 2, local_z, 15 - local_x,
                                           make_block<MAKE_KV("half", "upper")>(
                                               "minecraft:tall_grass"));
                        } else if (info.vegetation > 0.6) {
                            chunk.setBlock(local_y + 1, local_z, 15 - local_x,
                                           make_block("minecraft:short_grass"));
                        }
                    } else if (absolute_y > info.height - 5) {
                        chunk.setBlock(local_y, local_z, 15 - local_x,
                                       make_block("minecraft:dirt"));
                    } else {
                        break;
                    }
                }
            }
        }
    }
    for (int32_t local_z = 3; local_z < 13; local_z++) {
        for (int32_t local_x = 3; local_x < 13; local_x++) {
            FlatInfo info = chunk.get_flat_info(local_x, local_z);

            float starting_height = info.height + 15 + 64; // because local

            bool cold = info.temperature < 0.4f;
            int32_t surface_height = surface_heights[local_x][local_z];

            // Generate a tree
            if (surface_height > 130 && info.continentalness > 0.55 &&
                info.continentalness < 0.9 && info.vegetation > 0.675) {

                float tree_size =
                    5 + 2. * std::min<float>((info.vegetation - 0.675) / 0.025f,
                                             1.f);

                // Leaves:
                for (int y = 0; y < 4; y++) {
                    for (int z = -2; z < 3; z++) {
                        for (int x = -2; x < 3; x++) {
                            if (z * z + x * x + y * y + y > tree_size)
                                continue;

                            chunk.setBlock(surface_height + y + 3, local_z + z,
                                           15 - local_x + x,
                                           make_block("minecraft:oak_leaves"));
                        }
                    }
                }

                // Trunk:
                for (int y = 1; y < 4; y++) {
                    chunk.setBlock(surface_height + y, local_z, 15 - local_x,
                                   make_block("minecraft:oak_log"));
                }
            }
        }
    }
};

std::vector<Chunk> ChunkGenerator::generate_all(int32_t region_x,
                                                int32_t region_z) {
    // Storing all 2d noise etc.
    thrust::device_vector<FlatInfo> flats(16 * 16 * 32 * 32);

    // Use a lambda to map from index to (x, z) and call
    // getBaseTerrainHeight
    thrust::transform(
        thrust::counting_iterator<uint32_t>(0),
        thrust::counting_iterator<uint32_t>(16 * 16 * 32 * 32), flats.begin(),
        [seed = seed, region_x, region_z] __device__(uint32_t idx) {
            // Array of [[chunk]] where chunk is locally indexed
            int32_t cell_id = idx / (16 * 16);

            int32_t cell_id_x = cell_id / 32;
            int32_t cell_id_z = cell_id % 32;

            int32_t local_x = idx % 16;
            int32_t local_z = (idx / 16) % 16;

            int32_t chunk_x = (region_x * 32 + cell_id_x) * 16;
            int32_t chunk_z = (region_z * 32 + cell_id_z) * 16;

            return get_flat_info(seed, chunk_x + local_x, chunk_z + local_z);
        });

    thrust::device_vector<VolumetricInfo> volumetrics(16 * 16 * 16 * 24 * 32 *
                                                      32);

    thrust::transform(
        thrust::counting_iterator<int32_t>(0),
        thrust::counting_iterator<int32_t>(16 * 16 * 32 * 32 * 24 * 16),
        volumetrics.begin(),
        [seed = seed, region_x, region_z] __device__(int32_t idx) {
            // 16 x 16 x 16 blocks in subchunk
            // 24 subchunks in chunk
            // 32 x 32 chunks in region

            int32_t local_x = idx % 16;
            int32_t local_z = (idx / 16) % 16;
            int32_t local_y = (idx / 16 / 16) % 16;
            int32_t subchunk_id = (idx / (16 * 16 * 16)) % 24;
            int32_t cell_id = idx / (16 * 16 * 16 * 24);

            int32_t cell_id_x = cell_id / 32;
            int32_t cell_id_z = cell_id % 32;

            int32_t chunk_x = (region_x * 32 + cell_id_x) * 16;
            int32_t chunk_z = (region_z * 32 + cell_id_z) * 16;

            int32_t x = chunk_x + local_x;
            int32_t y = subchunk_id * 16 - 64 + local_y;
            int32_t z = chunk_z + local_z;

            auto density = noise(seed + 4, x, y, z, 0.1f, 2);
            bool cave = cave_noise(seed + 5, x, y, z);

            return VolumetricInfo{density, cave};
        });

    thrust::device_vector<ChunkSmol *> chunk_smols(32 * 32);
    std::vector<Chunk> chunks;
    chunks.reserve(32 * 32);

    for (auto cell_id = 0; cell_id < 32 * 32; cell_id++) {
        int32_t cell_id_x = cell_id / 32;
        int32_t cell_id_z = cell_id % 32;

        int32_t x = (region_x * 32 + cell_id_x) * 16;
        int32_t z = (region_z * 32 + cell_id_z) * 16;

        auto chunk = &chunks.emplace_back(x, z);
        chunk_smols[cell_id] = chunk->chunk_smols.get();
    }

    hipDeviceSynchronize();

    ChunkSmol **all_chunks = thrust::raw_pointer_cast(chunk_smols.data());
    FlatInfo *all_flats = thrust::raw_pointer_cast(flats.data());
    VolumetricInfo *all_volumetrics =
        thrust::raw_pointer_cast(volumetrics.data());

    thrust::for_each(thrust::counting_iterator<uint32_t>(0),
                     thrust::counting_iterator<uint32_t>(24 * 32 * 32),
                     [seed = seed, region_x, region_z, all_chunks, all_flats,
                      all_volumetrics] __device__(const uint32_t &smol_idx) {
                         const uint32_t i_ch = smol_idx % 24;
                         const uint32_t cell_id = smol_idx / 24;

                         ChunkSmol *raw_chunks = all_chunks[cell_id];

                         // Placement new to construct the ChunkSmol object
                         new (raw_chunks + i_ch) ChunkSmol();

                         int32_t cell_id_x = cell_id / 32;
                         int32_t cell_id_z = cell_id % 32;

                         int32_t chunk_x = (region_x * 32 + cell_id_x) * 16;
                         int32_t chunk_z = (region_z * 32 + cell_id_z) * 16;

                         // Generate this smol chunk
                         generateSmolChunk(
                             raw_chunks + i_ch, seed, chunk_x, i_ch * 16,
                             chunk_z, all_flats + cell_id * 16 * 16,
                             all_volumetrics + smol_idx * 16 * 16 * 16);
                     });

    hipDeviceSynchronize();

    thrust::for_each(thrust::counting_iterator<uint32_t>(0),
                     thrust::counting_iterator<uint32_t>(32 * 32),
                     [seed = seed, region_x, region_z, all_chunks,
                      all_flats] __device__(const uint32_t &cell_id) {
                         ChunkSmol *raw_chunks = all_chunks[cell_id];

                         int32_t cell_id_x = cell_id / 32;
                         int32_t cell_id_z = cell_id % 32;

                         int32_t chunk_x = (region_x * 32 + cell_id_x) * 16;
                         int32_t chunk_z = (region_z * 32 + cell_id_z) * 16;

                         ChunkWrapper wrapper(raw_chunks,
                                              all_flats + cell_id * 16 * 16,
                                              chunk_x, chunk_z);
                         replaceSurface(wrapper, seed);
                     });

    return chunks;
}
